#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>
//unsigned int atomicAdd(unsigned int* address, unsigned int val) {};


#define saes_data(w) {\
    w(0x63), w(0x7c), w(0x77), w(0x7b), w(0xf2), w(0x6b), w(0x6f), w(0xc5),\
    w(0x30), w(0x01), w(0x67), w(0x2b), w(0xfe), w(0xd7), w(0xab), w(0x76),\
    w(0xca), w(0x82), w(0xc9), w(0x7d), w(0xfa), w(0x59), w(0x47), w(0xf0),\
    w(0xad), w(0xd4), w(0xa2), w(0xaf), w(0x9c), w(0xa4), w(0x72), w(0xc0),\
    w(0xb7), w(0xfd), w(0x93), w(0x26), w(0x36), w(0x3f), w(0xf7), w(0xcc),\
    w(0x34), w(0xa5), w(0xe5), w(0xf1), w(0x71), w(0xd8), w(0x31), w(0x15),\
    w(0x04), w(0xc7), w(0x23), w(0xc3), w(0x18), w(0x96), w(0x05), w(0x9a),\
    w(0x07), w(0x12), w(0x80), w(0xe2), w(0xeb), w(0x27), w(0xb2), w(0x75),\
    w(0x09), w(0x83), w(0x2c), w(0x1a), w(0x1b), w(0x6e), w(0x5a), w(0xa0),\
    w(0x52), w(0x3b), w(0xd6), w(0xb3), w(0x29), w(0xe3), w(0x2f), w(0x84),\
    w(0x53), w(0xd1), w(0x00), w(0xed), w(0x20), w(0xfc), w(0xb1), w(0x5b),\
    w(0x6a), w(0xcb), w(0xbe), w(0x39), w(0x4a), w(0x4c), w(0x58), w(0xcf),\
    w(0xd0), w(0xef), w(0xaa), w(0xfb), w(0x43), w(0x4d), w(0x33), w(0x85),\
    w(0x45), w(0xf9), w(0x02), w(0x7f), w(0x50), w(0x3c), w(0x9f), w(0xa8),\
    w(0x51), w(0xa3), w(0x40), w(0x8f), w(0x92), w(0x9d), w(0x38), w(0xf5),\
    w(0xbc), w(0xb6), w(0xda), w(0x21), w(0x10), w(0xff), w(0xf3), w(0xd2),\
    w(0xcd), w(0x0c), w(0x13), w(0xec), w(0x5f), w(0x97), w(0x44), w(0x17),\
    w(0xc4), w(0xa7), w(0x7e), w(0x3d), w(0x64), w(0x5d), w(0x19), w(0x73),\
    w(0x60), w(0x81), w(0x4f), w(0xdc), w(0x22), w(0x2a), w(0x90), w(0x88),\
    w(0x46), w(0xee), w(0xb8), w(0x14), w(0xde), w(0x5e), w(0x0b), w(0xdb),\
    w(0xe0), w(0x32), w(0x3a), w(0x0a), w(0x49), w(0x06), w(0x24), w(0x5c),\
    w(0xc2), w(0xd3), w(0xac), w(0x62), w(0x91), w(0x95), w(0xe4), w(0x79),\
    w(0xe7), w(0xc8), w(0x37), w(0x6d), w(0x8d), w(0xd5), w(0x4e), w(0xa9),\
    w(0x6c), w(0x56), w(0xf4), w(0xea), w(0x65), w(0x7a), w(0xae), w(0x08),\
    w(0xba), w(0x78), w(0x25), w(0x2e), w(0x1c), w(0xa6), w(0xb4), w(0xc6),\
    w(0xe8), w(0xdd), w(0x74), w(0x1f), w(0x4b), w(0xbd), w(0x8b), w(0x8a),\
    w(0x70), w(0x3e), w(0xb5), w(0x66), w(0x48), w(0x03), w(0xf6), w(0x0e),\
    w(0x61), w(0x35), w(0x57), w(0xb9), w(0x86), w(0xc1), w(0x1d), w(0x9e),\
    w(0xe1), w(0xf8), w(0x98), w(0x11), w(0x69), w(0xd9), w(0x8e), w(0x94),\
    w(0x9b), w(0x1e), w(0x87), w(0xe9), w(0xce), w(0x55), w(0x28), w(0xdf),\
    w(0x8c), w(0xa1), w(0x89), w(0x0d), w(0xbf), w(0xe6), w(0x42), w(0x68),\
    w(0x41), w(0x99), w(0x2d), w(0x0f), w(0xb0), w(0x54), w(0xbb), w(0x16) }

#define SAES_WPOLY           0x011b

#define saes_b2w(b0, b1, b2, b3) (((uint32_t)(b3) << 24) | \
    ((uint32_t)(b2) << 16) | ((uint32_t)(b1) << 8) | (b0))

#define saes_f2(x)   ((x<<1) ^ (((x>>7) & 1) * SAES_WPOLY))
#define saes_f3(x)   (saes_f2(x) ^ x)
#define saes_h0(x)   (x)

#define saes_u0(p)   saes_b2w(saes_f2(p),          p,          p, saes_f3(p))
#define saes_u1(p)   saes_b2w(saes_f3(p), saes_f2(p),          p,          p)
#define saes_u2(p)   saes_b2w(         p, saes_f3(p), saes_f2(p),          p)
#define saes_u3(p)   saes_b2w(         p,          p, saes_f3(p), saes_f2(p))

static  __constant__  uint32_t saes_table[4][256] = { saes_data(saes_u0), saes_data(saes_u1), saes_data(saes_u2), saes_data(saes_u3) };


#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 64


#define AES2_EMU(s0, s1, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s0, &rc[rci + 2],sharedMemory1); \
  aesenc(&s1, &rc[rci + 3],sharedMemory1);

#define AES4(s0, s1, s2, s3, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s2, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 3],sharedMemory1); \
  aesenc(&s0, &rc[rci + 4], sharedMemory1); \
  aesenc(&s1, &rc[rci + 5], sharedMemory1); \
  aesenc(&s2, &rc[rci + 6], sharedMemory1); \
  aesenc(&s3, &rc[rci + 7], sharedMemory1);


#define AES4_LAST(s3, rci) \
  aesenc(&s3, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 6], sharedMemory1); \


#define TRUNCSTORE(out, s4) \
  *(uint32_t*)(out + 28) = s4.y;

#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);

#define MIX4_LASTBUT1(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); 

typedef struct case_struct {
	uint32_t *case_array[8];
} main_case_s;

__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id);


__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint4 * d_key_input, uint4 * d_mid, uint32_t *d_fix_r, uint32_t *d_fix_rex);
__global__ void verus_gpu_final(uint32_t threads, uint32_t startNonce, uint32_t *resNonce, uint4 * d_key_input, const  uint4 * d_mid);
__global__ void verus_extra_gpu_prepare(const uint32_t threads, uint4 * d_key_input);
__global__ void verus_extra_gpu_fix(const uint32_t threads, uint4 * d_key_input, uint32_t *d_fix_r, uint32_t *d_fix_rex);
__global__ void verus_case_0(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_4(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_8(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_c(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_10(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_14(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_18(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);
__global__ void verus_case_1c(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest);


static uint32_t *d_nonces[MAX_GPUS];
static uint4 *d_long_keys[MAX_GPUS];

static uint4 *d_mid[MAX_GPUS];
static main_case_s main_case_struct_p[MAX_GPUS];
static main_case_s main_case_dest[MAX_GPUS];
static uint32_t *d_case_totals[MAX_GPUS];
__device__ __constant__ uint4 vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint4 blockhash_half[4];
__device__ __constant__ uint4 blockhash_half2[4];
__device__ __constant__ uint32_t ptarget[8];

static hipStream_t streams[MAX_GPUS][8];
__host__
void verus_init(int thr_id, uint32_t throughput)
{
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(verus_gpu_hash), hipFuncCachePreferL1);
	//	printf("start of verus_init\n");
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMalloc(&d_long_keys[thr_id], throughput * VERUS_KEY_SIZE));
	CUDA_SAFE_CALL(hipMalloc(&d_mid[thr_id], throughput * 16));

	for (int i = 0; i < 8; i++)
	{
		CUDA_SAFE_CALL(hipMalloc(&main_case_struct_p[thr_id].case_array[i], throughput * sizeof(uint32_t)));
	}
	for (int i = 0; i < 8; i++)
	{
		CUDA_SAFE_CALL(hipMalloc(&main_case_dest[thr_id].case_array[i], throughput * sizeof(uint32_t)));
	}

	CUDA_SAFE_CALL(hipMalloc(&d_case_totals[thr_id], sizeof(uint32_t) * 8));
	//	printf("send of verus_init\n");
	for (int i = 0; i < 8; i++)
		hipStreamCreate(&streams[thr_id][i]);
};

__host__
void verus_setBlock(uint8_t blockf[64], uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput)
{

	//	printf("start of veruse_setblocks\n");
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half2), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	
	for(int i=16;i<32;i++)
	blockf[i] = blockf[i] ^ blockf[i+32];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey), (void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	dim3 grid2(throughput);
	//verus_extra_gpu_prepare << <grid2, 128 >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	
	//	printf("end of veruse_setblocks\n");main_case_struct_p[thr_id]
};

#define THREADS2 128

__global__
void copythings(uint32_t threads, main_case_s d_main_case, main_case_s d_main_case_dest)
{

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {
#pragma unroll 8
		for (int i = 0; i < 8; i++)
			d_main_case_dest.case_array[i][thread] = d_main_case.case_array[i][thread];

	}


}



__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces, uint32_t startcase)
{
	//printf("start of verus)hash\n");

	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	const uint32_t threadsperblock = THREADS;
	const uint32_t threadsperblock256 = 256;

	dim3 grid_final((threads + threadsperblock256 - 1) / threadsperblock256);
	dim3 grid_fix(threads);
	dim3 block(threadsperblock);

	uint32_t case_totals[8];
	startcase >>= 2;
	verus_extra_gpu_prepare << <grid_fix, 128 >> > (0, d_long_keys[thr_id]);
	//printf("startcase = %x, \t threads = %d\n", startcase, threads);
	for (int i = 0; i < 8; i++) {

		if (i == startcase)
			case_totals[i] = threads;
		else
			case_totals[i] = 0;


	}
	//CUDA_SAFE_CALL(hipMemcpy(d_case_totals[thr_id], case_totals, 8 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	main_case_s *case_pointer[2];
	case_pointer[0] = &main_case_struct_p[thr_id];
	case_pointer[1] = &main_case_dest[thr_id];
	int b, c;
	for (int i = 0; i < 32; i++)
	{
		b = i & 1;
		c = !(i & 1);
		CUDA_SAFE_CALL(hipMemset(d_case_totals[thr_id], 0, 8 * sizeof(uint32_t)));
		//hipDeviceSynchronize();

		if (case_totals[5] > 0) {

			dim3 grid5((case_totals[5] + threadsperblock - 1) / threadsperblock);
			verus_case_14 << <grid5, block, 0, streams[thr_id][0] >> > (case_totals[5], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[0] > 0) {
			//	printf("got to case 0, totals = %d, startnonce %d \n", case_totals[0], startNonce);

			dim3 grid0((case_totals[0] + threadsperblock - 1) / threadsperblock);
			verus_case_0 << <grid0, block, 0, streams[thr_id][1] >> > (case_totals[0], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[1] > 0) {
			//printf("got to case 1, totals = %d\n", case_totals[1]);
			dim3 grid1((case_totals[1] + threadsperblock - 1) / threadsperblock);
			verus_case_4 << <grid1, block, 0, streams[thr_id][2] >> > (case_totals[1], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[2] > 0) {
			//	printf("got to case 2, totals = %d\n", case_totals[2]);
			dim3 grid2((case_totals[2] + threadsperblock - 1) / threadsperblock);
			verus_case_8 << <grid2, block, 0, streams[thr_id][3] >> > (case_totals[2], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[3] > 0) {

			dim3 grid3((case_totals[3] + threadsperblock - 1) / threadsperblock);
			verus_case_c << <grid3, block, 0, streams[thr_id][4] >> > (case_totals[3], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[4] > 0) {

			dim3 grid4((case_totals[4] + threadsperblock - 1) / threadsperblock);
			verus_case_10 << <grid4, block, 0, streams[thr_id][5] >> > (case_totals[4], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}



		if (case_totals[6] > 0) {

			dim3 grid6((case_totals[6] + threadsperblock - 1) / threadsperblock);
			verus_case_18 << <grid6, block, 0, streams[thr_id][6] >> > (case_totals[6], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}

		if (case_totals[7] > 0) {

			dim3 grid7((case_totals[7] + threadsperblock - 1) / threadsperblock);
			verus_case_1c << <grid7, block, 0, streams[thr_id][7] >> > (case_totals[7], startNonce, d_long_keys[thr_id], d_mid[thr_id], 
				i, case_pointer[b][0], d_case_totals[thr_id], case_pointer[c][0]);
		}


		if (i < 31) 
			CUDA_SAFE_CALL(hipMemcpy(case_totals, d_case_totals[thr_id], 8 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	}

	//	exit(0);
	//	verus_gpu_hash << <grid, block >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]);
	verus_gpu_final << <grid_final, 256 >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], d_mid[thr_id]);
	//verus_extra_gpu_fix << <grid_fix, 32 >> > (0, d_long_keys[thr_id], d_fix_rand[thr_id], d_fix_randex[thr_id]); //setup global mem with lots of keys	
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	//exit(0);
};
__device__ __forceinline__
uint32_t xor3x(uint32_t a, uint32_t b, uint32_t c) {
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b), "r"(c)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

__device__  __forceinline__  uint4 _mm_xor_si128_emu(uint4 a, uint4 b)
{
	uint4 result;
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.x) : "r"(a.x), "r"(b.x));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.y) : "r"(a.y), "r"(b.y));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.z) : "r"(a.z), "r"(b.z));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(result.w) : "r"(a.w), "r"(b.w));
	return result;


}


__device__  __forceinline__  uint32_t XT4(uint32_t b)
{
	uint32_t tmp1, tmp2, tmp3;

	tmp1 = (b << 1) & 0xfefefefe;
	tmp2 = (b >> 7) & 0x1010101;
	asm("mul.lo.u32 %0, %1, 27; ": "=r"(tmp3) : "r"(tmp2));
	asm("xor.b32 %0, %1, %2; // xor1" : "=r"(tmp2) : "r"(tmp1), "r"(tmp3));

	return tmp2;
}

__device__  uint4 _mm_clmulepi64_si128_emu(uint4 ai, uint4 bi)
{
	uint64_t a = ((uint64_t*)&ai)[0]; // (0xffffffffull & ai.x) | ((0x00000000ffffffffull & ai.y) << 32);//+ (imm & 1));

	uint64_t b = ((uint64_t*)&bi)[1]; // (0xffffffffull & bi.z) | ((0x00000000ffffffffull & bi.w) << 32);

									  //	uint8_t  i; 
									  //	uint2 u[8];
	uint64_t r[2]; //uint4 r;
				   //uint2 tmp;

	if (__popcll(a) > __popcll(b)) {

		a = b; b = ((uint64_t*)&ai)[0];
	}
	r[0] = 0; r[1] = 0;

	uint64_t w = a; int counter = 0; int first;

	while ((first = __clzll(w) + 1) != 65) {
		w <<= (first);
		counter += (first);

		r[0] ^= b << (64 - counter);
		r[1] ^= b >> ((counter));
	};

	return ((uint4*)&r)[0];
}

__device__  __forceinline__ uint4 _mm_clmulepi64_si128_emu2(uint4 ai)
{
	uint64_t a = ((uint64_t*)&ai)[1];

	//uint64_t b = 27 ;
	uint8_t  i; //window size s = 4,
				//uint64_t two_s = 16; //2^s
				//uint64_t smask = 15; //s 15 
	uint8_t u[8];
	uint4 r;
	uint64_t tmp;
	//Precomputation

	//#pragma unroll
	u[0] = 0;  //000 x b
	u[1] = 27;  //001 x b
	u[2] = 54; // u[1] << 1; //010 x b
	u[3] = 45;  //011 x b
	u[4] = 108; //100 x b
	u[5] = 119;  //101 x b
	u[6] = 90; //110 x b
	u[7] = 65;  //111 x b
				//Multiply
	((uint64_t*)&r)[0] = u[a & 7]; //first window only affects lower word

	r.z = r.w = 0;
	//#pragma unroll
	for (i = 3; i < 64; i += 3) {
		tmp = u[a >> i & 7];
		r.x ^= (tmp << i) & 0xffffffff;
		r.y ^= ((tmp << i) & 0xffffffff00000000) >> 32;
		//	((uint64_t*)&r)[1] ^= tmp >> (64 - i);
		r.z ^= (tmp >> (64 - i)) & 0xffffffff;
		r.w ^= ((tmp >> (64 - i)) & 0xffffffff00000000) >> 32;
	}

	return r;
}

#define _mm_load_si128_emu(p) (*(uint4*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);


__device__  void _mm_unpackboth_epi32_emu(uint4 &a, uint4 &b)
{
	uint64_t value;

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(a.z), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(a.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.x), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(b.x) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.z), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.y), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.y) : "l"(value));
}


__device__  __forceinline__ uint4 _mm_unpacklo_epi32_emu(uint4 a, uint4 b)
{

	//uint4 t;

	//	t.x = a.x;
	a.z = a.y;
	a.y = b.x;
	a.w = b.y;
	return a;
}

__device__  __forceinline__ uint4 _nop(uint4 a, uint4 b)
{

	return a;
}

__device__  __forceinline__ uint4 _mm_unpackhi_epi32_emu(uint4 a, uint4 b)
{

	//uint4 t;
	b.x = a.z;
	b.y = b.z;
	b.z = a.w;
	//t.w = b.w;

	return b;
}

__device__ uint32_t get_byte(uint32_t a, uint32_t start_bit)
{
	uint32_t result;
	asm("bfe.u32 %0, %1, %2, 8;" : "=r"(result) : "r"(a), "r"(start_bit));
	return result;
}

__device__   __forceinline__ void aesenc(uint4 * __restrict__ ptr, const uint4 * __restrict__ key, uint32_t * __restrict__ t)
{
	//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

	//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))
	uint32_t x0 = ptr[0].x;
	uint32_t x1 = ptr[0].y;
	uint32_t x2 = ptr[0].z;
	uint32_t x3 = ptr[0].w;

	uint32_t y0 = t[get_byte(x0, 0)];
	uint32_t y1 = t[get_byte(x1, 0)];
	uint32_t y2 = t[get_byte(x2, 0)];
	uint32_t y3 = t[get_byte(x3, 0)];
	t += 256;

	y0 ^= t[get_byte(x1, 8)];
	y1 ^= t[get_byte(x2, 8)];
	y2 ^= t[get_byte(x3, 8)];
	y3 ^= t[get_byte(x0, 8)];
	t += 256;

	y0 ^= t[get_byte(x2, 16)];
	y1 ^= t[get_byte(x3, 16)];
	y2 ^= t[get_byte(x0, 16)];
	y3 ^= t[get_byte(x1, 16)];
	t += 256;

	y0 ^= t[get_byte(x3, 24)];
	y1 ^= t[get_byte(x0, 24)];
	y2 ^= t[get_byte(x1, 24)];
	y3 ^= t[get_byte(x2, 24)];

	ptr[0].x = y0 ^ key[0].x;
	ptr[0].y = y1 ^ key[0].y;
	ptr[0].z = y2 ^ key[0].z;
	ptr[0].w = y3 ^ key[0].w;

}


__device__  __forceinline__ uint4 _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint4 result = { 0 };
	result.x = lo;

	return result;
}
__device__  __forceinline__ uint4 _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint4 result = { 0 };
	((uint64_t *)&result)[0] = lo;
	//((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint4 _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint4 result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}

__device__ __forceinline__ uint4 _mm_shuffle_epi8_emu(uint4 b)
{
	uint4 result = { 0 };
	const uint4 M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	const uint4 Q = { 0x80808080, 0x80808080, 0x80808080, 0x80808080 };
	const uint4 W = b & Q;


#pragma unroll
	for (int i = 0; i < 8; i++)
	{
		if (!((uint8_t *)&W)[i])
		{
			((uint8_t *)&result)[i] = ((uint8_t *)&M)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}



__device__  __forceinline__ uint4 _mm_srli_si128_emu(uint4 input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint4 temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
	((uint64_t*)&temp)[1] = 0;


	return temp;
}



__device__  uint4 _mm_mulhrs_epi16_emu(uint4 _a, uint4 _b)
{
	int16_t result[8];

	int32_t po;
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma nounroll
	for (int i = 0; i < 8; i++)
	{
		asm("mad.lo.s32 %0, %1, %2, 16384; ": "=r"(po) : "r"((int32_t)a[i]), "r"((int32_t)b[i]));

		result[i] = po >> 15;
		//	result[i] = (int16_t)((((int32_t)(a[i]) * (int32_t)(b[i])) + 0x4000) >> 15);

	}
	return *(uint4 *)result;
}


__device__   __forceinline__  uint32_t haraka512_port_keyed(uint4 * __restrict__  in, uint4 * __restrict__  rc, uint32_t * __restrict__  sharedMemory1)
{
	uint4 s1, s2, s3, s4, tmp;

	s1 = in[0];
	s2 = in[1];
	s3 = in[2];
	s4 = in[3];

	AES4(s1, s2, s3, s4, 0);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 8);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 16);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 24);
	MIX4_LASTBUT1(s1, s2, s3, s4);


	AES4_LAST(s3, 32);

	return s3.z ^ in[3].y;

}

__device__   __forceinline__ uint64_t precompReduction64(uint4 A) {


	//static const uint4 M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	// const uint4 tmp = { 27 };
	// A.z = 0;
	//tmp.x = 27u;
	uint4 Q2 = _mm_clmulepi64_si128_emu2(A);
	uint4 Q3 = _mm_shuffle_epi8_emu(_mm_srli_si128_emu(Q2, 8));

	//uint4 Q4 = _mm_xor_si128_emu(Q2, A);
	uint4 final;
	final.x = xor3(A.x, Q2.x, Q3.x);
	final.y = xor3(A.y, Q2.y, Q3.y);

	return _mm_cvtsi128_si64_emu(final);/// WARNING: HIGH 64 BITS SHOULD BE ASSUMED TO CONTAIN GARBAGE
}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_0(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];

		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[0][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4
		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];
		//printf("iteration %d got past%d acc %08x  %08x  %08x\n",iteration, nonce, randomsource[123].x, prand.x, prandex.x);

		uint4 temp1 = prandex;

		uint4 temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
		uint4 add1 = _mm_xor_si128_emu(temp1, temp2);
		uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1);
		acc = _mm_xor_si128_emu(clprod1, acc);

		uint4 tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
		uint4 tempa2 = _mm_xor_si128_emu(tempa1, temp1);

		uint4 temp12 = prand;
		prand = tempa2;

		uint4 temp22 = _mm_load_si128_emu(pbuf);
		uint4 add12 = _mm_xor_si128_emu(temp12, temp22);
		uint4 clprod12 = _mm_clmulepi64_si128_emu(add12, add12);
		acc = _mm_xor_si128_emu(clprod12, acc);

		uint4 tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		uint4 tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prandex = tempb2;

		//  //d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}


__global__ __launch_bounds__(THREADS, 1)
void verus_case_4(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[1][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

		uint4 temp1 = prand;
		uint4 temp2 = _mm_load_si128_emu(pbuf);
		uint4 add1 = _mm_xor_si128_emu(temp1, temp2);
		uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1);
		acc = _mm_xor_si128_emu(clprod1, acc);
		uint4 clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2);
		acc = _mm_xor_si128_emu(clprod2, acc);

		uint4 tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
		uint4 tempa2 = _mm_xor_si128_emu(tempa1, temp1);

		uint4 temp12 = prandex;
		prandex = tempa2;

		uint4 temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
		uint4 add12 = _mm_xor_si128_emu(temp12, temp22);
		acc = _mm_xor_si128_emu(add12, acc);

		uint4 tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		uint4 tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_8(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[2][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

		uint4 temp1 = prandex;
		uint4 temp2 = _mm_load_si128_emu(pbuf);
		uint4 add1 = _mm_xor_si128_emu(temp1, temp2);
		acc = _mm_xor_si128_emu(add1, acc);

		uint4 tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
		uint4 tempa2 = _mm_xor_si128_emu(tempa1, temp1);

		uint4 temp12 = prand;
		prand = tempa2;

		uint4 temp22 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
		uint4 add12 = _mm_xor_si128_emu(temp12, temp22);
		uint4 clprod12 = _mm_clmulepi64_si128_emu(add12, add12);
		acc = _mm_xor_si128_emu(clprod12, acc);
		uint4 clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22);
		acc = _mm_xor_si128_emu(clprod22, acc);

		uint4 tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		uint4 tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prandex = tempb2;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_c(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[3][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];
		//if (nonce == 1)
		//	printf("[i]=%d \t acc = %08x, prand_idx = %d\t, prandex_idx = %d \t prand %08x, prandex %08x case 3\n", iteration, acc.x, prand_idx, prandex_idx, prand.x, prandex.x);

		const uint4 temp1 = prand;
		const uint4 temp2 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
		const uint4 add1 = _mm_xor_si128_emu(temp1, temp2);

		// cannot be zero here
		const int32_t divisor = ((uint32_t*)&selector)[0];

		acc = _mm_xor_si128_emu(add1, acc);

		int64_t dividend = _mm_cvtsi128_si64_emu(acc);
		int64_t tmpmod = dividend % divisor;
		const uint4 modulo = _mm_cvtsi32_si128_emu(tmpmod);
		acc = _mm_xor_si128_emu(modulo, acc);

		const uint4 tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
		const uint4 tempa2 = _mm_xor_si128_emu(tempa1, temp1);
		dividend &= 1;
		if (dividend)
		{
			const uint4 temp12 = prandex;
			prandex = tempa2;

			const uint4 temp22 = _mm_load_si128_emu(pbuf);
			const uint4 add12 = _mm_xor_si128_emu(temp12, temp22);
			const uint4 clprod12 = _mm_clmulepi64_si128_emu(add12, add12);
			acc = _mm_xor_si128_emu(clprod12, acc);
			const uint4 clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22);
			acc = _mm_xor_si128_emu(clprod22, acc);

			const uint4 tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
			const uint4 tempb2 = _mm_xor_si128_emu(tempb1, temp12);
			prand = tempb2;
		}
		else
		{
			const uint4 tempb3 = prandex;
			prandex = tempa2;
			prand = tempb3;
		}

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;

		//	if (nonce == 0)
		//		printf("[i]=%d \t acc = %08x, prand_idx = %d\t, prandex_idx = %d \t prand %08x, prandex %08x case 3\n", iteration, acc.x, prand_idx, prandex_idx, prand.x, prandex.x);


	}

}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_10(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	__shared__ uint32_t shared_m[4][256];

	for (int i = threadIdx.x; i < 256; i += blockDim.x) {

		shared_m[0][i] = saes_table[0][i];
		shared_m[1][i] = saes_table[1][i];
		shared_m[2][i] = saes_table[2][i];
		shared_m[3][i] = saes_table[3][i];
	}
	__syncthreads();

	uint32_t *sharedMemory1 = shared_m[0];

	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[4][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

		uint4 *rc = &randomsource[prand_idx];

		uint4 tmp;

		uint4 temp1 = _mm_load_si128_emu(pbuf - (((selector & 1) << 1) - 1));
		uint4 temp2 = _mm_load_si128_emu(pbuf);

		AES2_EMU(temp1, temp2, 0);
		MIX2_EMU(temp1, temp2);


		AES2_EMU(temp1, temp2, 4);
		MIX2_EMU(temp1, temp2);

		AES2_EMU(temp1, temp2, 8);
		MIX2_EMU(temp1, temp2);


		acc = _mm_xor_si128_emu(temp1, acc);
		acc = _mm_xor_si128_emu(temp2, acc);

		const uint4 tempa1 = prand;
		const uint4 tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1); selector = _mm_cvtsi128_si64_emu(acc);
		const uint4 tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

		const uint4 tempa4 = prandex;
		prandex = tempa3;
		prand = tempa4;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_14(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	__shared__ uint32_t shared_m[4][256];

	for (int i = threadIdx.x; i < 256; i += blockDim.x) {

		shared_m[0][i] = saes_table[0][i];
		shared_m[1][i] = saes_table[1][i];
		shared_m[2][i] = saes_table[2][i];
		shared_m[3][i] = saes_table[3][i];
	}
	__syncthreads();

	uint32_t *sharedMemory1 = shared_m[0];

	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[5][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

		// we'll just call this one the monkins loop, inspired by Chris
		const uint4 *buftmp = pbuf - (((selector & 1) << 1) - 1);
		uint4 tmp; // used by MIX2

		uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
		uint4 *rc = &randomsource[prand_idx];


		uint64_t aesround = 0;
		uint4 onekey;
		uint64_t loop_c;


		for (int i = 0; i<8; i++)
		{
			if (rounds <= 8) {
				loop_c = selector & ((uint64_t)0x10000000 << rounds);
				if (loop_c)
				{
					onekey = _mm_load_si128_emu(rc++);
					const uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
					const uint4 add1 = _mm_xor_si128_emu(onekey, temp2);
					const uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1);
					acc = _mm_xor_si128_emu(clprod1, acc);
				}
				else
				{
					onekey = _mm_load_si128_emu(rc++);
					uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);

					const uint64_t roundidx = aesround++ << 2;
					AES2_EMU(onekey, temp2, roundidx);

					MIX2_EMU(onekey, temp2);

					acc = _mm_xor_si128_emu(onekey, acc);
					acc = _mm_xor_si128_emu(temp2, acc);

				}

				(rounds--);
			}
		}

		const uint4 tempa1 = (prand);
		const uint4 tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
		const uint4 tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

		const uint4 tempa4 = (prandex);
		prandex = tempa3;
		prand = tempa4;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}

__global__ __launch_bounds__(THREADS, 1)
void verus_case_18(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];


		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[6][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4

		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];

		// we'll just call this one the monkins loop, inspired by Chris
		const uint4 *buftmp = pbuf - (((selector & 1) << 1) - 1);


		uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
		uint4 *rc = &randomsource[prand_idx];

		uint4 onekey;
		uint64_t loop_c;

		for (int i = 0; i<8; i++)
		{
			if (rounds <= 8) {
				loop_c = selector & ((uint64_t)0x10000000 << rounds);
				if (loop_c)
				{
					onekey = _mm_load_si128_emu(rc++);
					const uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
					const uint4 add1 = _mm_xor_si128_emu(onekey, temp2);

					const int32_t divisor = (uint32_t)selector;
					const int64_t dividend = ((int64_t*)&add1)[0];
					uint4 modulo = { 0 }; ((int32_t*)&modulo)[0] = (dividend % divisor);
					acc = modulo ^ acc;

				}
				else
				{
					onekey = _mm_load_si128_emu(rc++);
					uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);
					uint4 add1 = (onekey^ temp2);
					uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1);
					uint4 clprod2 = _mm_mulhrs_epi16_emu(acc, clprod1);
					acc = clprod2^ acc;
				}
			}
			(rounds--);
		}

		const uint4 tempa3 = (prandex);
		const uint4 tempa4 = _mm_xor_si128_emu(tempa3, acc);
		prandex = tempa4;
		prand = onekey;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;
	}

}


__global__ __launch_bounds__(THREADS, 1)
void verus_case_1c(const uint32_t threads, const uint32_t startNonce, uint4 * __restrict__ d_key_input, uint4 * __restrict__ d_mid,
	 uint32_t iteration, main_case_s  d_main_case, uint32_t *totals, main_case_s  d_case_dest)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
		uint4 s[4];
		s[0] = blockhash_half[0];
		s[1] = blockhash_half[1];
		s[2] = blockhash_half[2];
		s[3] = blockhash_half[3];

		uint32_t nonce = (iteration == 0) ? thread : d_main_case.case_array[7][thread]; //first iteration nonce is just 0..throughput
		uint4 acc = (iteration == 0) ? vkey[513] : d_mid[nonce];
		uint4 *randomsource = &d_key_input[VERUS_KEY_SIZE128 * nonce];
		s[2].x = nonce + startNonce;
		s[0] = s[0] ^ s[2];
		//s[1] = s[1] ^ s[3];
		// divide key mask by 32 from bytes to uint4



		uint16_t prand_idx, prandex_idx;
		uint64_t selector = _mm_cvtsi128_si64_emu(acc);
		uint4 const *pbuf = s + (selector & 3);
		uint4 prand;
		uint4 prandex;
		prand_idx = ((acc.x >> 5) & 511);
		prandex_idx = ((acc.y) & 511);

		prand = randomsource[prand_idx];
		prandex = randomsource[prandex_idx];
		//	if(nonce ==1)
		//		printf("[i]=%d \t acc = %08x, prand_idx = %d\t, prandex_idx = %d \t prand %08x, prandex %08x case 7\n", iteration, acc.x, prand_idx, prandex_idx, prand.x, prandex.x);

		//	if (nonce == 0)
		//		printf("pbuf %08x%08x%08x%08x\n",s[0].x, s[1].x, s[2].x, s[3].x);

		const uint4 temp1 = _mm_load_si128_emu(pbuf);
		const uint4 temp2 = (prandex);
		const uint4 add1 = _mm_xor_si128_emu(temp1, temp2);
		const uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1);
		acc = _mm_xor_si128_emu(clprod1, acc);
		//	if (nonce == 0)
		//		printf("acc = %08x",acc.x);


		const uint4 tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
		const uint4 tempa2 = _mm_xor_si128_emu(tempa1, temp2);
		const uint4 tempa3 = (prand);


		prand = tempa2;

		acc = _mm_xor_si128_emu(tempa3, acc);
		//	if (nonce == 0)
		//		printf("[i]=%d \t acc = %08x, prand_idx = %d\t, prandex_idx = %d \t prand %08x, prandex %08x\n", iteration, acc.x, prand_idx, prandex_idx, prand.x, prandex.x);

		const uint4 tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
		const uint4 tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
		prandex = tempb2;

		//d_fix_r[(nonce * 32) + iteration] = prand_idx;
		//d_fix_rex[(nonce * 32) + iteration] = prandex_idx;
		randomsource[prand_idx] = prand;
		randomsource[prandex_idx] = prandex;
		d_mid[nonce] = acc;
		selector = acc.x & 0x1c;
		selector >>= 2;
		uint32_t location = atomicAdd(&totals[selector], 1);
		d_case_dest.case_array[selector][location] = nonce;

		//	if (nonce == 0)
		//	printf("[i]=%d \t acc = %08x, prand_idx = %d\t, prandex_idx = %d \t prand %08x, prandex %08x\n", iteration, acc.x, prand_idx, prandex_idx, prand.x, prandex.x);

	}

}

__global__ __launch_bounds__(256, 1)
void verus_gpu_final(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint4 * __restrict__  d_key_input, const uint4 * __restrict__ d_mid)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint4 mid = d_mid[thread];
	mid.x ^= 0x00010000;

	uint64_t acc = precompReduction64(mid);
	//uint4 wizz = d_mid[thread];

	const uint32_t nounce = startNonce + thread;
	uint32_t hash;

	uint4 s[4];
	__shared__ uint32_t sharedMemory1[4][256];
	//	sharedMemory1[threadIdx.x] = sbox[threadIdx.x];// copy sbox to shared mem
	sharedMemory1[0][threadIdx.x] = saes_table[0][threadIdx.x];// copy sbox to shared mem

	sharedMemory1[1][threadIdx.x] = saes_table[1][threadIdx.x];// copy sbox to shared mem

	sharedMemory1[2][threadIdx.x] = saes_table[2][threadIdx.x];// copy sbox to shared mem


	sharedMemory1[3][threadIdx.x] = saes_table[3][threadIdx.x];// copy sbox to shared mem



	s[0] = blockhash_half2[0];
	s[1] = blockhash_half2[1];
	s[2] = blockhash_half2[2];
	s[3] = blockhash_half2[3];
	__syncthreads();

	((uint32_t *)&s)[8] = nounce;
	memcpy(((uint8_t*)&s) + 47, &acc, 8);
	memcpy(((uint8_t*)&s) + 55, &acc, 8);
	memcpy(((uint8_t*)&s) + 63, &acc, 1);
	//uint64_t mask = 8191 >> 4;
	acc &= 511;


	hash = haraka512_port_keyed(s, (&d_key_input[VERUS_KEY_SIZE128 * thread] + acc), sharedMemory1[0]);
	if (hash < ptarget[7]) {

		resNonce[0] = nounce;

	}


};

__global__ __launch_bounds__(128, 1)
void verus_extra_gpu_prepare(const uint32_t threads, uint4 * d_key_input)
{

	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x] = vkey[threadIdx.x];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 128] = vkey[threadIdx.x + 128];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 256] = vkey[threadIdx.x + 256];
	d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 384] = vkey[threadIdx.x + 384];
	if (threadIdx.x < 40)
		d_key_input[(blockIdx.x * VERUS_KEY_SIZE128) + threadIdx.x + 512] = vkey[threadIdx.x + 512];

}

